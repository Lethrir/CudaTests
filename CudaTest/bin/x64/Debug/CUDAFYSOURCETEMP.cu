
#include <hip/hip_runtime.h>
struct ProgramTestClass
{
	__device__  ProgramTestClass()
	{
	}
	int A;
	int B;
	int C;
};


// CudaTest.Program
extern "C" __global__ void doTheThing( ProgramTestClass* tests, int testsLen0);
// CudaTest.Program
extern "C" __global__ void addArrays( int* a, int aLen0,  int* b, int bLen0,  int* c, int cLen0);
// CudaTest.Program
extern "C" __global__ void findPrimes( int* toCheck, int toCheckLen0,  int* results, int resultsLen0);
// CudaTest.Program
__device__ int isPrime(int a);

// CudaTest.Program
extern "C" __global__ void doTheThing( ProgramTestClass* tests, int testsLen0)
{
	int x = blockIdx.x;
	if (x < 1000000)
	{
		ProgramTestClass testClass = tests[(x)];
		testClass.C = testClass.A + testClass.B;
	}
}
// CudaTest.Program
extern "C" __global__ void addArrays( int* a, int aLen0,  int* b, int bLen0,  int* c, int cLen0)
{
	int x = blockIdx.x;
	if (x < 1000000)
	{
		c[(x)] = a[(x)] + b[(x)];
	}
}
// CudaTest.Program
extern "C" __global__ void findPrimes( int* toCheck, int toCheckLen0,  int* results, int resultsLen0)
{
	int x = blockIdx.x;
	if (x < 1000000)
	{
		results[(x)] = isPrime(toCheck[(x)]);
	}
}
// CudaTest.Program
__device__ int isPrime(int a)
{
	int result;
	if (a == 1 || a == 2)
	{
		result = 1;
	}
	else
	{
		int num = a % 2;
		if (num == 0)
		{
			result = 0;
		}
		else
		{
			int num2 = a / 2;
			for (int num3 = 3; num3 <= num2; num3++)
			{
				if (a % num3 == 0)
				{
					result = 0;
					return result;
				}
			}
			result = 1;
		}
	}
	return result;
}
